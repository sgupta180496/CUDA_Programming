
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly,int degree,int n,float* array) 
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<n)
  	{
		float result=0.0;
  		float exponent=1.0;
  		for(int x=0;x<=degree;++x)
  		{
  			result+=exponent*poly[x];
  			exponent*=array[index];
  		}
  		array[index]=result;
  	}
}

int main(int argc, char* argv[]) 
{
  	if(argc<3) 
  	{
  		std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
  		return -1;
  	}

	int n=atoi(argv[1]); 
	int degree=atoi(argv[2]);
	int nbiter=1;
  	float* array=new float[n];
  	float* poly=new float[degree+1];
  	for(int i=0;i<n;++i)
	{
  		array[i]=1.;
	}

  	for(int i=0;i<degree+1;++i)
	{
  		poly[i]=1.;
	}

  	float *d_array,*d_poly;

	//start calculating time
  	std::chrono::time_point<std::chrono::system_clock> start_time,end_time;
  	start_time = std::chrono::system_clock::now();

  	hipMalloc(&d_array,n*sizeof(float));
  	hipMalloc(&d_poly,(degree+1)*sizeof(float));

  	hipMemcpy(d_array,array,n*sizeof(float),hipMemcpyHostToDevice);
  	hipMemcpy(d_poly,poly,(degree+1)*sizeof(float),hipMemcpyHostToDevice);

  	polynomial_expansion<<<(n+255)/256, 256>>>(d_poly,degree,n,d_array);
  	hipMemcpy(array,d_array,n*sizeof(float),hipMemcpyDeviceToHost);

  	hipFree(d_array);
  	hipFree(d_poly);

  	hipDeviceSynchronize();
	{
	    	bool correct=true;
	    	int ind;
		for(int i=0;i<n;++i) 
		{
			if(fabs(array[i]-(degree+1))>0.01) 
			{
				correct=false;
				ind=i;
			}
		}
	    	if(!correct)
	    	std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
	}
	// calculate and print time
  	end_time=std::chrono::system_clock::now();
  	std::chrono::duration<double> elapsed_time=(end_time-start_time)/nbiter;
  	std::cout<<n<<" "<<degree<<" "<<elapsed_time.count()<<std::endl;
	
	// free arrays
  	delete[] array;
  	delete[] poly;

  	return 0;
}


